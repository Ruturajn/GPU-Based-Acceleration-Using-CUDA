#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <stdlib.h>
#include <opencv2/highgui.hpp>
#include <opencv2/imgproc.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;
using namespace cv;

#define kernel_width 9
#define THREADSx     8
#define THREADSy     8
#define BLOCK_PADDED (kernel_width + THREADSx)

/*****************************Convolution Kernel******************************/
__global__ void ImgBlurKernel(unsigned char* d_img, int num_rows, int num_cols,
                              float* d_filter, int filter_width, 
                              unsigned char* d_res){
    
  // Calculating the row and column indices
  int row = ((threadIdx.y * 2)  + (blockDim.y * blockIdx.y));
  int col = ((threadIdx.x * 2) + (blockDim.x * blockIdx.x));

  // Declaring shared memory
  __shared__ unsigned char temp_img[(BLOCK_PADDED - 1)*(BLOCK_PADDED - 1)*3];
      
        // Case 1
        int start_row = row - filter_width/2;
        int start_col = col - filter_width/2;

        int img_idx = (start_col);
        int img_idy = (start_row);

        int temp_idx = (threadIdx.x * 2);
        int temp_idy = (threadIdx.y * 2);

        int temp_id_f = ((temp_idy * (BLOCK_PADDED-1) + temp_idx)*3);
        int img_id_f = ((img_idy*num_cols + img_idx)*3);

        int x_n = 0, y_n = 0;

        if (start_row < 0 || start_col < 0 || start_row > num_rows -1 || start_col > num_cols -1){

        y_n = (start_row < 0) ? 0 : ((start_row >= num_rows) ? (num_rows-1) : (start_row));
        x_n = (start_col < 0) ? 0 : ((start_col >= num_cols) ? (num_cols-1) : (start_col));
    

        img_id_f = ((y_n*num_cols + x_n)*3); 

        temp_img[temp_id_f] = d_img[img_id_f];
        temp_img[temp_id_f + 1] = d_img[img_id_f + 1];
        temp_img[temp_id_f + 2] = d_img[img_id_f + 2];

        }
        else{
            temp_img[temp_id_f] = d_img[img_id_f];
            temp_img[temp_id_f + 1] = d_img[img_id_f + 1];
            temp_img[temp_id_f + 2] = d_img[img_id_f + 2];
        }

        // Case 2
        img_idx = start_col;
        img_idy = start_row + 1;

        temp_idx = (threadIdx.x * 2);
        temp_idy = (threadIdx.y * 2) + 1;

        temp_id_f = ((temp_idy * (BLOCK_PADDED-1) + temp_idx)*3);
        img_id_f = ((img_idy*num_cols + img_idx)*3);
        
        if ((start_row+1) > num_rows-1 || start_col < 0 || (start_row+1) < 0 || start_col > num_cols -1){

        y_n = (start_row < 0) ? 0 : ((start_row >= num_rows) ? (num_rows-1) : (start_row+1));
        x_n = (start_col < 0) ? 0 : ((start_col >= num_cols) ? (num_cols-1) : (start_col));

        img_id_f = ((y_n*num_cols + x_n)*3); 

        temp_img[temp_id_f] = d_img[img_id_f];
        temp_img[temp_id_f + 1] = d_img[img_id_f + 1];
        temp_img[temp_id_f + 2] = d_img[img_id_f + 2];

        }
        else{
            temp_img[temp_id_f] = d_img[img_id_f];
            temp_img[temp_id_f + 1] = d_img[img_id_f + 1];
            temp_img[temp_id_f + 2] = d_img[img_id_f + 2];
        }

        // Case 3
        img_idx = start_col + 1;
        img_idy = start_row;

        temp_idx = (threadIdx.x * 2) + 1;
        temp_idy = (threadIdx.y * 2);

        temp_id_f = ((temp_idy * (BLOCK_PADDED-1) + temp_idx)*3);
        img_id_f = ((img_idy*num_cols + img_idx)*3);

        if ((start_col+1) > num_cols -1 || start_row < 0 || (start_col+1) < 0 || start_row > num_rows - 1){

        y_n = (start_row < 0) ? 0 : ((start_row >= num_rows) ? (num_rows-1) : start_row);
        x_n = (start_col < 0) ? 0 : ((start_col >= num_cols) ? (num_cols-1) : start_col+1);

        img_id_f = ((y_n*num_cols + x_n)*3); 

        temp_img[temp_id_f] = d_img[img_id_f];
        temp_img[temp_id_f + 1] = d_img[img_id_f + 1];
        temp_img[temp_id_f + 2] = d_img[img_id_f + 2];

        }
        else{
            temp_img[temp_id_f] = d_img[img_id_f];
            temp_img[temp_id_f + 1] = d_img[img_id_f + 1];
            temp_img[temp_id_f + 2] = d_img[img_id_f + 2];
        }

        // Case 4
        img_idx = start_col + 1;
        img_idy = start_row + 1;

        temp_idx = (threadIdx.x * 2) + 1;
        temp_idy = (threadIdx.y * 2) + 1;

        temp_id_f = ((temp_idy * (BLOCK_PADDED-1) + temp_idx)*3);
        img_id_f = ((img_idy*num_cols + img_idx)*3);

        if ((start_row+1) > num_rows -1 || (start_col+1) > num_cols - 1 || (start_col+1) < 0 || (start_row+1) < 0){
 
        y_n = (start_row < 0) ? 0 : ((start_row >= num_rows) ? (num_rows-1) : start_row+1);
        x_n = (start_col < 0) ? 0 : ((start_col >= num_cols) ? (num_cols-1) : start_col+1);

        img_id_f = ((y_n*num_cols + x_n)*3); 

        temp_img[temp_id_f] = d_img[img_id_f];
        temp_img[temp_id_f + 1] = d_img[img_id_f + 1];
        temp_img[temp_id_f + 2] = d_img[img_id_f + 2];

        }
        else{
            temp_img[temp_id_f] = d_img[img_id_f];
            temp_img[temp_id_f + 1] = d_img[img_id_f + 1];
            temp_img[temp_id_f + 2] = d_img[img_id_f + 2];
        }


    // Barrier for all threads
    __syncthreads();


    //*************************************************************************

    //Performing convolution

    // Initializing temporary product variable

    col = (threadIdx.x + (blockDim.x*blockIdx.x));
    row = (threadIdx.y + (blockDim.y*blockIdx.y));

    if (row < num_rows && col < num_cols){

        float prod_r = 0.0f, prod_g = 0.0f, prod_b = 0.0f;

        float img_val_r = 0, img_val_g = 0, img_val_b = 0;

        int temp_id_conv = 0;
        float filter_val = 0.0f;
        
        for (int i=0;i<filter_width;i++){
            for (int j=0;j<filter_width;j++){

                temp_id_conv = (((threadIdx.y + i)*(BLOCK_PADDED-1) + (threadIdx.x + j))*3);
                
                filter_val = d_filter[i*filter_width + j];
                
                img_val_r = float(temp_img[temp_id_conv]);
                img_val_g = float(temp_img[temp_id_conv + 1]);
                img_val_b = float(temp_img[temp_id_conv + 2]);

                prod_r += (img_val_r * filter_val);
                prod_g += (img_val_g * filter_val);
                prod_b += (img_val_b * filter_val);
            }
        }


        int idx = ((row*num_cols + col)*3);

        d_res[idx] = prod_r;
        d_res[idx + 1] = prod_g;
        d_res[idx + 2] = prod_b;

    }
}

/*************************Pre-Processing Function*****************************/
void ImgBlur(Mat& h_img, Mat& h_res, float* filter, int filter_width){

        // Create events to time the kernel
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        float exec_time;
        
        // Calculating the size of the frame for memory
        size_t img_size = (h_img.rows * h_img.cols * h_img.channels());

        // Calculating the size of the filter
        size_t filter_size = (filter_width * filter_width * sizeof(float));

        // Defining Device variables
        unsigned char *d_frame, *d_res;
        float* d_filter;

        // Allocating memory
        hipMalloc((void**)&d_frame, img_size);
        hipMalloc((void**)&d_res, img_size);
        hipMalloc((void**)&d_filter, filter_size);

        // Copying frame to device
        hipMemcpy(d_frame, h_img.ptr(), img_size, hipMemcpyHostToDevice);

        // Copying filter from host to device
        hipMemcpy(d_filter, filter, filter_size, hipMemcpyHostToDevice);

        // Configuring threads and blocks
        int BLOCKSx = (h_img.cols + THREADSx -1) / THREADSx;
        int BLOCKSy = (h_img.rows + THREADSy -1) / THREADSy;

        dim3 threads(THREADSx, THREADSy);
        dim3 blocks(BLOCKSx, BLOCKSy);

        // Launching the kernel
        hipEventRecord(start);
        ImgBlurKernel<<<blocks, threads>>>(d_frame, h_img.rows, h_img.cols,
                                           d_filter, filter_width, d_res);

        hipDeviceSynchronize();

        hipError_t err = hipGetLastError();

        if (err != hipSuccess){
            // print the CUDA error message and exit
            printf("CUDA error: %s\n", hipGetErrorString(err));
            exit(-1);
        }

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&exec_time, start, stop);

        // Copying the results back to host
        hipMemcpy(h_res.ptr(), d_res, img_size, hipMemcpyDeviceToHost);

        std::cout << "Time required to execute the kernel is : " << exec_time << endl;

        // Free the allocated memory
        hipFree(d_frame);
        hipFree(d_res);
        hipFree(d_filter);

        return;
}

/************************Function to create Filter****************************/
void create_filter(float* h_filter){
      
  //now create the filter that they will use
  const int blurKernelWidth = kernel_width;
  const float blurKernelSigma = 2.;

  float filterSum = 0.f; //for normalization

  for (int r = -blurKernelWidth/2; r <= blurKernelWidth/2; ++r) {
    for (int c = -blurKernelWidth/2; c <= blurKernelWidth/2; ++c) {
      float filterValue = expf( -(float)(c * c + r * r) / (2.f * blurKernelSigma * blurKernelSigma));
      (h_filter)[(r + blurKernelWidth/2) * blurKernelWidth + c + blurKernelWidth/2] = filterValue;
      filterSum += filterValue;
    }
  }

  float normalizationFactor = 1.f / filterSum;

  for (int r = -blurKernelWidth/2; r <= blurKernelWidth/2; ++r) {
    for (int c = -blurKernelWidth/2; c <= blurKernelWidth/2; ++c) {
      (h_filter)[(r + blurKernelWidth/2) * blurKernelWidth + c + blurKernelWidth/2] *= normalizationFactor;
    }
  }
}



int main(void){

    // Reading the image

    //Mat h_img = imread("../Images/index.jpeg", 1);
    //Mat h_img = imread("../Images/cinque_terre_small.jpg", 1);
    //Mat h_img = imread("/usr/share/backgrounds/brad-huchteman-stone-mountain.jpg", 1);
    //Mat h_img = imread("../Images/138728.jpg", 1);
    //Mat h_img = imread("../Images/2040735.jpg", 1);
    Mat h_img = imread("../Images/UI-Sidewalk-640x480.jpg", 1);


    //resize(h_img, h_img, Size(280, 184));
    //resize(h_img, h_img, Size(560, 320));
    cout << h_img.cols << " || " << h_img.rows << endl;

    // Resizing the image
    if (h_img.cols % 8 != 0 && h_img.rows % 8 != 0){
        resize(h_img, h_img, Size((h_img.cols/THREADSx + 1)*THREADSx, (h_img.rows/THREADSy + 1)*THREADSy));
    }
    cout << h_img.cols << " || " << h_img.rows << endl;

    // Create the output image
    Mat h_res(h_img.rows, h_img.cols, CV_8UC3);

    // Calculate the filter size and allocate memory for it
    size_t filter_size = (kernel_width * kernel_width * sizeof(float));
    float* h_filter = (float*)malloc(filter_size);

    // Call the create_filter to fill the filter
    create_filter(h_filter);

    // Call the pre-processing function, that further calls the kernel
    ImgBlur(h_img, h_res, h_filter, kernel_width);

    // Calculate the difference based on the in-built function in OpenCV
    Mat ref_img;
    cv::GaussianBlur(h_img, ref_img, Size2i(kernel_width, kernel_width), 0);
    Mat diff_opencv = abs(ref_img - h_res);

    // Show the results
    cv::imshow("Original", h_img);
    cv::imshow("Blurred", h_res);
    cv::imshow("CV-Blur", ref_img);
    cv::imshow("diff-CV", diff_opencv);

    // Saving Images to disk
    imwrite("../Images/Blurred_Shared_mem_final_fruits.png", h_res);
    imwrite("../Images/Difference_Image_Shared_mem.png", diff_opencv);

    // Wait for any keystrokes
    cv::waitKey(0);

    // Free the memory allocated to the filter
    delete[] h_filter;

    return 0;
}
